//
// Created by DanielSun on 9/7/2021.
//

#include "TrainData.cuh"

void TrainData::AllocateDataOnDeviceMem() {

    (void) hipMalloc(reinterpret_cast<void**>(&dataG), sizeof(MatrixOperations::Matrix2d));
    (void) hipMalloc(reinterpret_cast<void**>(&dataG->elements), DATA_INPUT_SIZE_1D * sizeof(float));
    (void) hipMalloc(reinterpret_cast<void**>(&labelG), sizeof(MatrixOperations::Matrix2d));
    (void) hipMalloc(reinterpret_cast<void**>(&labelG->elements), DATA_OUTPUT_SIZE_1D * sizeof(float));

    // dist, src, size, mode
    (void) hipMemcpy(dataG,data, sizeof(MatrixOperations::Matrix2d), hipMemcpyHostToDevice);
    (void) hipMemcpy(dataG->elements, data->elements, data->colcount * data->rowcount * sizeof(float), hipMemcpyHostToDevice);
    (void) hipMemcpy(labelG, label, sizeof(MatrixOperations::Matrix2d), hipMemcpyHostToDevice);
    (void) hipMemcpy(labelG->elements, label->elements, label->rowcount * label -> colcount * sizeof(float), hipMemcpyHostToDevice);
}

void TrainData::FreeDataOnDeviceMem() const {
    (void) hipFree(dataG->elements);
    (void) hipFree(dataG);
}
