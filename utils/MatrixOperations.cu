#include "hip/hip_runtime.h"
//
// Created by DanielSun on 9/14/2021.
//
#include "../NetConfig.cuh"
#include "MatrixOperations.cuh"
using namespace std;

//global functions: operations

__global__ void setElement(MatrixOperations::Matrix2d *mat1, int row, int col, float value){
    MatrixOperations::setElement2D(mat1,row, col, value);
}

__global__ void allocateElements(MatrixOperations::Matrix2d *mat1,int row, int col){
    mat1->rowcount = row;
    mat1->colcount = col;
    (void)hipMalloc(reinterpret_cast<void**>(&mat1->elements),5*sizeof(float));
}
//random number fill (0-1)
__global__ void allocRandom(long seed, MatrixOperations::Matrix2d *mat1){
    hiprandStateXORWOW_t state;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(row*col*seed,0,0,&state);
    MatrixOperations::setElement2D(mat1, row, col, static_cast<float>((hiprand_uniform(&state)*2.0F) -1.0F));
}

//zero fill
__global__ void allocZero( MatrixOperations::Matrix2d *mat1){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    MatrixOperations::setElement2D(mat1, row, col, 0.0F);
}

//cross product
__global__ void crossProduct2D(MatrixOperations::Matrix2d *mat1, MatrixOperations::Matrix2d *mat2,
                               MatrixOperations::Matrix2d *resultBuffer) {
    float currentValue = 0.0;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = 0; i < mat1->colcount; ++i) {
        currentValue += MatrixOperations::getElement2D(mat1, row, i) *
                        MatrixOperations::getElement2D(mat2, i, col);
    }
    MatrixOperations::setElement2D(resultBuffer, row, col, currentValue);
}

__global__ void constantProduct2D(MatrixOperations::Matrix2d *mat1, float k){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = k*MatrixOperations::getElement2D(mat1, row, col);
    MatrixOperations::setElement2D(mat1, row, col, currentValue);
}

__global__ void transit2D(MatrixOperations::Matrix2d *input, MatrixOperations::Matrix2d *result){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = MatrixOperations::getElement2D(input, row, col);
    MatrixOperations::setElement2D(result, col, row, currentValue);
}


__global__ void addition2D(MatrixOperations::Matrix2d *input,MatrixOperations::Matrix2d *mat2){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = MatrixOperations::getElement2D(input, row, col);
    currentValue += MatrixOperations::getElement2D(mat2, row, col);
    MatrixOperations::setElement2D(input, row, col, currentValue);
}

__global__ void addition2D(MatrixOperations::Matrix2d *mat1, MatrixOperations::Matrix2d *mat2,
                           MatrixOperations::Matrix2d *output){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = MatrixOperations::getElement2D(mat1, row, col);
    currentValue += MatrixOperations::getElement2D(mat2, row, col);
    MatrixOperations::setElement2D(output, row, col, currentValue);
}

__global__ void subtraction2D(MatrixOperations::Matrix2d *mat1, MatrixOperations::Matrix2d *mat2){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = MatrixOperations::getElement2D(mat1, row, col);
    currentValue -= MatrixOperations::getElement2D(mat2, row, col);
    MatrixOperations::setElement2D(mat1, row, col, currentValue);
}
__global__ void subtraction2D(MatrixOperations::Matrix2d *mat1, MatrixOperations::Matrix2d *mat2,
                              MatrixOperations::Matrix2d *result){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = MatrixOperations::getElement2D(mat1, row, col);
    currentValue -= MatrixOperations::getElement2D(mat2, row, col);
    MatrixOperations::setElement2D(mat1, row, col, currentValue);
}

__global__ void exponential2D(MatrixOperations::Matrix2d *mat1, float value){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = MatrixOperations::getElement2D(mat1, row, col);
    currentValue = pow(currentValue, value);
    MatrixOperations::setElement2D(mat1, row, col, currentValue);
}

__global__ void exponential2D(MatrixOperations::Matrix2d *mat1, float value, MatrixOperations::Matrix2d *result){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = MatrixOperations::getElement2D(mat1, row, col);
    currentValue = pow(currentValue, value);
    MatrixOperations::setElement2D(result, row, col, currentValue);
}

__global__ void hadmardProduct(MatrixOperations::Matrix2d *mat1, MatrixOperations::Matrix2d *mat2){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = MatrixOperations::getElement2D(mat1, row, col);
    currentValue *= MatrixOperations::getElement2D(mat2,row,col);
    MatrixOperations::setElement2D(mat1, row, col, currentValue);
}

__global__ void derivativeSigmoid(MatrixOperations::Matrix2d *input){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = MatrixOperations::getElement2D(input, row, col);
    currentValue =  (1.0F/(1.0F + exp(-currentValue))) * (1.0F-(1.0F/(1.0F + exp(-currentValue))));
    MatrixOperations::setElement2D(input, row, col, currentValue);
}

__global__ void activationSigmoid(MatrixOperations::Matrix2d *input){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = MatrixOperations::getElement2D(input, row, col);
    currentValue = 1.0F/(1.0F + exp(-currentValue));  //sigmoid function
    MatrixOperations::setElement2D(input, row, col, currentValue);
}

__global__ void activationSigmoid(MatrixOperations::Matrix2d *input, MatrixOperations::Matrix2d *result){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = MatrixOperations::getElement2D(input, row, col);
    currentValue = 1.0F/(1.0F + exp(-currentValue));  //sigmoid function
    MatrixOperations::setElement2D(result, row, col, currentValue);
}

__global__ void MatCpy(MatrixOperations::Matrix2d *source, MatrixOperations::Matrix2d *dist){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float currentValue = MatrixOperations::getElement2D(source, row, col);
    MatrixOperations::setElement2D(dist, row, col, currentValue);
}

__device__ float MatrixOperations::getElement2D(MatrixOperations::Matrix2d *source, int row, int col) {
    return source->elements[row * source->colcount + col];
}

__device__ void MatrixOperations::setElement2D(MatrixOperations::Matrix2d *source, int row, int col, float value) {
    source->elements[row * source->colcount + col] = value;
}

void MatrixOperations::callCrossProduct2D( MatrixOperations::Matrix2d *mat1,
                                           MatrixOperations::Matrix2d *mat2, MatrixOperations::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    //condition
    if (mat1->colcount != mat2->rowcount || mat1->colcount != result -> colcount
        || mat2->rowcount != result -> rowcount){
        LogUtils::logErr("CrossProduct2D : Dimensional Mismatch");
        throw exception("CrossProduct2D : Dimensional Mismatch");
    }


    (void)crossProduct2D<<<gridSize, CUDA_BLOCK_SIZE>>> (mat1, mat2, result);
    (void)hipDeviceSynchronize();
}

void MatrixOperations::callAddition( MatrixOperations::Matrix2d *mat1,
                                     MatrixOperations::Matrix2d *mat2) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    if (mat1 -> colcount != mat2 -> colcount || mat1 -> rowcount != mat2 -> rowcount){
        LogUtils::logErr("Addition2D : Dimensional Mismatch");
        throw exception("Addition2D : Dimensional Mismatch");
    }
    (void)addition2D<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, mat2);
    (void)hipDeviceSynchronize();
}

void MatrixOperations::callAddition(MatrixOperations::Matrix2d *mat1,
                                    MatrixOperations::Matrix2d *mat2, MatrixOperations::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    if (mat1 -> colcount != mat2 -> colcount || mat1 -> rowcount != mat2 -> rowcount){
        LogUtils::logErr("Addition2D : Dimensional Mismatch");
        throw exception("Addition2D : Dimensional Mismatch");
    }
    (void)addition2D<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, mat2, result);
    (void)hipDeviceSynchronize();
}


void MatrixOperations::callSigmoidActivation( MatrixOperations::Matrix2d *mat1) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    (void)activationSigmoid<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1);
    (void)hipDeviceSynchronize();
}

void MatrixOperations::callSigmoidActivation( MatrixOperations::Matrix2d *mat1,
                                              MatrixOperations::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    (void)activationSigmoid<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, result);
    (void)hipDeviceSynchronize();
}


void MatrixOperations::callSigmoidDerivative(MatrixOperations::Matrix2d *mat1) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    (void)derivativeSigmoid<<<gridSize,CUDA_BLOCK_SIZE>>>(mat1);
    (void)hipDeviceSynchronize();
}

void MatrixOperations::callHardmardProduct(MatrixOperations::Matrix2d *mat1,
                                           MatrixOperations::Matrix2d *mat2) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    (void) hadmardProduct<<<gridSize,CUDA_BLOCK_SIZE>>>(mat1, mat2);
    (void) hipDeviceSynchronize();
}

void MatrixOperations::callConstantProduct2D( MatrixOperations::Matrix2d *mat1, float k) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    (void) constantProduct2D<<<gridSize,CUDA_BLOCK_SIZE>>>(mat1, k);
    (void) hipDeviceSynchronize();
}

void MatrixOperations::callTransit2D( MatrixOperations::Matrix2d *mat1,
                                      MatrixOperations::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    if (mat1->colcount != result->rowcount || mat1->rowcount != result->colcount){
        LogUtils::logErr("Transit2D : Dimensional Mismatch");
        throw exception("Transit2D : Dimensional Mismatch");
    }
    (void) transit2D<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1,result);
    (void) hipDeviceSynchronize();
}

void MatrixOperations::callSubtraction(MatrixOperations::Matrix2d *mat1,
                                       MatrixOperations::Matrix2d *mat2) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    if (mat1 -> colcount != mat2 -> colcount || mat1 -> rowcount != mat2 -> rowcount){
        LogUtils::logErr("Addition2D : Dimensional Mismatch");
        throw exception("Addition2D : Dimensional Mismatch");
    }
    (void)addition2D<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, mat2);
    (void)hipDeviceSynchronize();
}

void MatrixOperations::callSubtraction( MatrixOperations::Matrix2d *mat1,
                                        MatrixOperations::Matrix2d *mat2, MatrixOperations::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    if (mat1 -> colcount != mat2 -> colcount || mat1 -> rowcount != mat2 -> rowcount){
        LogUtils::logErr("Addition2D : Dimensional Mismatch");
        throw exception("Addition2D : Dimensional Mismatch");
    }
    (void)addition2D<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, mat2, result);
    (void)hipDeviceSynchronize();
}

void MatrixOperations::callExponential(MatrixOperations::Matrix2d *mat1, float value) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    (void) exponential2D<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, value);
    (void)hipDeviceSynchronize();
}

void MatrixOperations::callExponential( MatrixOperations::Matrix2d *mat1, float value,
                                        MatrixOperations::Matrix2d *result) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    (void) exponential2D<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1, value,  result);
    (void) hipDeviceSynchronize();
}

void MatrixOperations::callMatCopy( MatrixOperations::Matrix2d *src,
                                    MatrixOperations::Matrix2d *dist) {
    dim3 gridSize = dim3((src->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (src->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    (void) MatCpy<<<gridSize,  CUDA_BLOCK_SIZE>>>(src,dist);
    (void) hipDeviceSynchronize();
}



void MatrixOperations::callAllocElement(MatrixOperations::Matrix2d *mat1, int row, int col) {
    (void) allocateElements<<<dim3(1,1), CUDA_BLOCK_SIZE>>>(mat1, row, col);
    (void) hipDeviceSynchronize();
}

void MatrixOperations::callAllocZero(MatrixOperations::Matrix2d *mat1) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    (void) allocZero<<<gridSize, CUDA_BLOCK_SIZE>>>(mat1);
    (void) hipDeviceSynchronize();
}

void MatrixOperations::callAllocRandom(MatrixOperations::Matrix2d *mat1) {
    dim3 gridSize = dim3((mat1->colcount + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x,
                         (mat1->rowcount + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y);
    (void) allocRandom<<<gridSize, CUDA_BLOCK_SIZE>>>(time(nullptr),mat1);
    (void) hipDeviceSynchronize();
}

void MatrixOperations::callSetElement(MatrixOperations::Matrix2d *mat1, int row, int col, float value) {
    (void) setElement<<<dim3(1,1),CUDA_BLOCK_SIZE>>>(mat1, row, col, value);
    (void)hipDeviceSynchronize();
}