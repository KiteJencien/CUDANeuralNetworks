//
// Created by DanielSun on 9/14/2021.
//

#include "DenseLayer.cuh"
#include "../NetConfig.cuh"

void DenseLayer::calculateActivations(MatrixOperations::Matrix2d *prevNodes)  {
    //z = w * a + b , a1 = sigmoid(z)
    MatrixOperations::callCrossProduct2D( this->weights, prevNodes,this->z);
    MatrixOperations::callAddition(this->z,this->bias);
    MatrixOperations::callSigmoidActivation(this->z, this->nodes);
}

// Error = (aL - y) *hadmard* sigmoidDerivative(z)
void DenseLayer::calculateErrorsAsOutput(MatrixOperations::Matrix2d *correctOut)  {
    MatrixOperations::callSigmoidDerivative( this->z);
    MatrixOperations::callSubtraction(this->z, correctOut, this->errors);
    //result will be stored in "errors"
    MatrixOperations::callHardmardProduct( this->errors, this->z);
}

//deltaWeight = ErrorL * (a(L-1))^T
//apply propagate : Weights - deltaWeights
void DenseLayer::propagateWeights(MatrixOperations::Matrix2d *prevActivations)  {
    MatrixOperations::Matrix2d *transit;
    (void) hipMallocManaged(reinterpret_cast<void**>(&transit), sizeof(MatrixOperations::Matrix2d));
    MatrixOperations::callAllocElementD(transit, 1, prevActivations->rowcount);
    MatrixOperations::callTransit2D( prevActivations, transit);
    MatrixOperations::callCrossProduct2D( this->errors, transit, this->weightDerivative);
    MatrixOperations::callConstantProduct2D( this->weightDerivative, LEARNING_RATE);
    MatrixOperations::callSubtraction( this->weights, this->weightDerivative);
    (void) hipFree(transit->elements);
    (void) hipFree(transit);
}

// E^l = (W^(l+1))^T * E^(l+1) <Hardmard> sigmoid'(z^l)
void DenseLayer::calculateErrors(DenseLayer *thisLayer, MatrixOperations::Matrix2d *nextLayerWeights, MatrixOperations::Matrix2d *nextLayerErr)  {
    MatrixOperations::Matrix2d *transitWeights;
    (void) hipMallocManaged(reinterpret_cast<void**>(&transitWeights), sizeof(MatrixOperations::Matrix2d));
    MatrixOperations::callAllocElementD(transitWeights, nextLayerWeights->colcount, nextLayerWeights->rowcount);
    MatrixOperations::callTransit2D( nextLayerWeights, transitWeights);
    MatrixOperations::callCrossProduct2D( transitWeights, nextLayerErr, thisLayer->errors);
    MatrixOperations::callSigmoidDerivative( thisLayer->z);
    MatrixOperations::callHardmardProduct( thisLayer->errors, thisLayer->z);
    (void) hipFree(transitWeights->elements);
    (void) hipFree(transitWeights);
}

void DenseLayer::rememberErrors() {
    MatrixOperations::callAddition( this->pastErrors, this->errors);
}

void DenseLayer::propagateBias() {
    MatrixOperations::callConstantProduct2D( this->errors, LEARNING_RATE);
    MatrixOperations::callSubtraction( this->bias, this->errors);
    MatrixOperations::callConstantProduct2D( this->errors, 1.0F/LEARNING_RATE);
}

void DenseLayer::applyErrors() {
    MatrixOperations::callConstantProduct2D( this->pastErrors, 1.0F/static_cast<float>(BATCH_SIZE));
    MatrixOperations::callMatCopy( this->pastErrors, this->errors);
    MatrixOperations::callAllocZero(this->pastErrors);
}

void DenseLayer::calculateErrors(Layer *in, Layer *next) {
    //do not call this method for layers that are not dense layers
    auto* thisLayer = dynamic_cast<DenseLayer*>(in);
    auto* nextLayer = dynamic_cast<DenseLayer*>(next);
    calculateErrors(thisLayer, nextLayer->weights, nextLayer->errors);
}







